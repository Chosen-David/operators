#include "../../../devices/cuda/common_cuda.h"
#include "pooling.cuh"

<<<<<<< HEAD
infiniopStatus_t pooling_nv_gpu(PoolingCudaDescriptor_t desc, void *y, void const *x, void *stream) {
    checkCudaError(hipSetDevice(desc->device_id));
    checkCudnnError(use_cudnn(desc->cudnn_handles_t, desc->device_id, (hipStream_t) stream,
=======
infiniopStatus_t pooling_nv_gpu(PoolingCudaDescriptor_t desc, void *workspace, uint64_t workspace_size, void *y, void const *x) {
    checkCudaError(hipSetDevice(desc->device_id));
    checkCudnnError(use_cudnn(desc->cudnn_handles_t, desc->device_id,
>>>>>>> 561f952 (Add Pooling (CUDA))
                              [&](hipdnnHandle_t handle) { return hipdnnPoolingForward(handle, desc->pool_desc,
                                                                                     &desc->alpha, desc->x_desc, x, &desc->beta,
                                                                                     desc->y_desc, y); }));
    return STATUS_SUCCESS;
}

infiniopStatus_t cudaPooling(PoolingCudaDescriptor_t desc,
<<<<<<< HEAD
                             void *workspace, uint64_t workspace_size,
                             void *y, void const *x, void *stream) {
    if (desc->dtype == F16 || desc->dtype == F32) {
        return pooling_nv_gpu(desc, y, x, stream);
=======
                             void *y, void const *x, void *stream) {

    if (desc->dtype == F16 || desc->dtype == F32) {
        return pooling_nv_gpu(desc, nullptr, 0, y, x);
>>>>>>> 561f952 (Add Pooling (CUDA))
    }
    return STATUS_BAD_TENSOR_DTYPE;
}
