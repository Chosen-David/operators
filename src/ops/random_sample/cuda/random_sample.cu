#include "hip/hip_runtime.h"
#include "../../../devices/cuda/common_cuda.h"
#include "../../utils.h"
#include "random_sample.cuh"
#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>

template<class T, int BLOCK_DIM>
__global__ void softmax(
    T *val_out,
    int topk,
    float temperature, int voc) {
    float sum_s = 0.0f;
    for (int i = threadIdx.x; i < topk; i += BLOCK_DIM) {
        sum_s += __expf(static_cast<float>(val_out[i] - val_out[0]) / temperature);
    }
    __shared__ float sum_inverse_total;

    typedef hipcub::BlockReduce<float, BLOCK_DIM> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    float block_sum = BlockReduce(temp_storage).Reduce(sum_s, hipcub::Sum());
    if (threadIdx.x == 0) {
        sum_inverse_total = __fdividef(1.0F, block_sum);//高精度除法
    }

    __syncthreads();
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < topk) {
        val_out[tid] = static_cast<T>(__expf(static_cast<float>(val_out[tid] - val_out[0]) / temperature) * sum_inverse_total);
    }
}

__global__ void index(int *key_in, int voc) {
    int ind = threadIdx.x + blockIdx.x * blockDim.x;
    if (ind < voc) {
        key_in[ind] = ind;
    }
}
template<class T>
__global__ void random_sample_kernel(int *result,
                                     T *val_out,
                                     float topp,
                                     int topk,
                                     int *key_out) {
    int end = 0;
    for (end = 0; end < topk; end++) {
        if (val_out[end] >= static_cast<T>(topp)) {
            break;
        }
    }
    if (end < topk - 1) {
        end += 1;
    } else {
        end = topk;
    }
    T randomVal = 0.75;
    randomVal *= val_out[end - 1];
    for (int i = 0; i < end; i++) {
        if (randomVal < val_out[i]) {
            result[0] = key_out[i];
            break;
        }
    }
}
void random_sample_nv_gpu_f16(RandomSampleCudaDescriptor_t desc, void *workspace, void *result,
                              void *probs,
                              float topp,
                              int topk,
                              float temperature,
                              void *stream) {
    int voc = desc->voc;
    //下面这段代码在排序


    half *val_out;
    hipMalloc((void **) &val_out, voc * sizeof(half));
    int *key_in, *key_out;
    hipMalloc((void **) &key_in, voc * sizeof(int));
    hipMalloc((void **) &key_out, voc * sizeof(int));
    index<<<(voc + 1023) / 1024, 1024, 0, (hipStream_t) stream>>>(key_in, voc);
    //下面开始计算workspace空间
    size_t size_radix_sort;
    hipcub::DeviceRadixSort::SortPairsDescending(
        nullptr, size_radix_sort,
        (half *) probs, val_out,
        key_in, key_out,
        voc, 0, sizeof(half) * 8, (hipStream_t) stream);
    size_t size_scan;
    hipcub::DeviceScan::InclusiveSum(
        nullptr, size_scan,
        val_out, val_out, voc,
        (hipStream_t) stream);
    //计算出workspace总共需要的字节数
    hipMalloc(&workspace, size_radix_sort + size_scan);
    hipcub::DeviceRadixSort::SortPairsDescending(
        workspace, size_radix_sort,
        (half *) probs, val_out,
        key_in, key_out,
        voc, 0, sizeof(half) * 8, (hipStream_t) stream);//该函数会把排序结果和对应索引保存在val_out和key_out上
    //排序结束，然后开始做softmax变换

    int BLOCK_DIM = 1024;
    int num_blocks = (voc + BLOCK_DIM - 1) / BLOCK_DIM;
    softmax<half, 1024><<<num_blocks, BLOCK_DIM, 0, (hipStream_t) stream>>>(val_out, topk,
                                                                             temperature, voc);


    hipcub::DeviceScan::InclusiveSum(
        workspace, size_scan,
        val_out, val_out, voc,
        (hipStream_t) stream);//该函数会实现scan功能不断累加结果
    random_sample_kernel<half><<<1, 1, 0, (hipStream_t) stream>>>((int *) result,
                                                                   val_out,
                                                                   topp,
                                                                   topk,
                                                                   key_out);
    hipFree(val_out);
    hipFree(key_in);
    hipFree(key_out);
}

infiniopStatus_t cudaRandomSample(RandomSampleCudaDescriptor_t desc,
                                  void *workspace,
                                  uint64_t workspace_size,
                                  void *result,
                                  void *probs,
                                  float topp,
                                  int topk,
                                  float temperature,
                                  void *stream) {
    if (hipSetDevice(desc->device_id) != hipSuccess) {
        return STATUS_BAD_DEVICE;
    }
    if (dtype_eq(desc->dtype, F16)) {
        random_sample_nv_gpu_f16(desc, workspace, result, probs, topp, topk, temperature, stream);
        return STATUS_SUCCESS;
    }

    return STATUS_BAD_TENSOR_DTYPE;
}
