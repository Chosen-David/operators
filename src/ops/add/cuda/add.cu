#include "hip/hip_runtime.h"
#include "../../../devices/cuda/common_cuda.h"
#include "../../utils.h"
#include "add.cuh"

template<typename T, int N>
struct vecN {
    T data[N];

    __device__ vecN operator+(const vecN<T, N> &other) const {
        vecN<T, N> result;
        for (int i = 0; i < N; ++i) {
            result.data[i] = data[i] + other.data[i];
        }
        return result;
    }

    __device__ const T &operator[](int i) const {
        return data[i];
    }
};

__device__ uint64_t getDstIndex(uint64_t flat_index, uint64_t ndim, int64_t const *src_strides, int64_t const *dst_strides) {
    uint64_t res = 0;
    for (uint64_t i = 0; i < ndim; ++i) {
        res += flat_index / src_strides[i] * dst_strides[i];
        flat_index %= src_strides[i];
    }
    return res;
}

template<typename Tdata, typename BTdata>
__global__ void add(
    Tdata *c,
    const Tdata *a,
    const Tdata *b,
    const int64_t *a_strides,
    const int64_t *b_strides,
    const int64_t *c_strides,
    uint64_t data_size,
    uint64_t ndim,
    uint64_t offset,
    bool broadcasted,
    unsigned pack_size) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

    if (idx < data_size) {
        if (broadcasted) {
            idx *= pack_size;
            auto a_ = reinterpret_cast<const BTdata *>(a);
            auto b_ = reinterpret_cast<const BTdata *>(b);
            auto c_ = reinterpret_cast<BTdata *>(c);
            for (size_t i = 0; i < pack_size; ++i) {
                auto a_idx = getDstIndex(idx + i, ndim, c_strides, a_strides);
                auto b_idx = getDstIndex(idx + i, ndim, c_strides, b_strides);
                c_[idx + i] = a_[a_idx] + b_[b_idx];
            }
            return;
        }
        c[idx] = a[idx] + b[idx];
    }
}

template<typename Tdata, typename BTdata>
void _add_nv_gpu(AddCudaDescriptor_t desc, Tdata *c, Tdata const *a, Tdata const *b, uint64_t data_size, uint64_t pack_size, uint64_t offset, void *stream) {
    if (data_size == 0) {
        return;
    }
    dim3 blockDims = dim3(std::min(static_cast<uint64_t>(MAX_THREADS_PER_BLOCK), data_size));
    dim3 gridDims = dim3(std::min(ROUND_UP_DIV(data_size, blockDims.x), desc->max_grid_size));
    uint64_t step = gridDims.x * blockDims.x;

    hipStream_t hip_stream = reinterpret_cast<hipStream_t>(stream);

    for (uint64_t i = 0; i < data_size; i += step) {
        add<Tdata, BTdata><<<gridDims, blockDims, 0, hip_stream>>>(
            c, a, b, desc->a_strides, desc->b_strides, desc->c_strides, offset + data_size, desc->ndim, offset + i, desc->broadcasted, pack_size);
    }
}

template<typename Tdata, typename TIdata>
void add_nv_gpu(AddCudaDescriptor_t desc, void *c, void const *a, void const *b, void *stream, uint64_t pack_size) {
    auto data_size = desc->c_data_size / pack_size;
    auto a_vec = reinterpret_cast<const Tdata *>(a);
    auto b_vec = reinterpret_cast<const Tdata *>(b);
    auto c_vec = reinterpret_cast<Tdata *>(c);
    _add_nv_gpu<Tdata, TIdata>(desc, c_vec, a_vec, b_vec, data_size, pack_size, 0, stream);

    auto remainder = desc->c_data_size % pack_size;
    auto a_ = reinterpret_cast<const TIdata *>(a);
    auto b_ = reinterpret_cast<const TIdata *>(b);
    auto c_ = reinterpret_cast<TIdata *>(c);
    _add_nv_gpu<TIdata, TIdata>(desc, c_, a_, b_, remainder, 1, data_size * pack_size, stream);
}

infiniopStatus_t cudaAdd(AddCudaDescriptor_t desc,
                         void *c, void const *a, void const *b,
                         void *stream) {
    checkCudaError(hipSetDevice(desc->device_id));
    if (desc->dtype == F16) {
        add_nv_gpu<vecN<half, 4>, half>(desc, c, a, b, stream, 4);
        return STATUS_SUCCESS;
    }
    if (desc->dtype == F32) {
        add_nv_gpu<vecN<float, 4>, float>(desc, c, a, b, stream, 4);
        return STATUS_SUCCESS;
    }
    return STATUS_BAD_TENSOR_DTYPE;
}
