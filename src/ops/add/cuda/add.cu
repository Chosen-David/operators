#include "../../../devices/cuda/common_cuda.h"
#include "../../utils.h"
#include "add.cuh"

void add_nv_gpu_f16(AddCudaDescriptor_t desc, void *c, void *a, void *b, void *stream) {
    // Create and set tensor descriptors for tensors a, b, and c
    hipdnnTensorDescriptor_t tensorDesc;
    checkCudnnError(hipdnnCreateTensorDescriptor(&tensorDesc));
    checkCudnnError(hipdnnSetTensorNdDescriptor(tensorDesc, HIPDNN_DATA_HALF, desc->ndim, desc->shape, desc->strides));

    hipdnnOpTensorDescriptor_t opDesc;
    checkCudnnError(hipdnnCreateOpTensorDescriptor(&opDesc));
    checkCudnnError(hipdnnSetOpTensorDescriptor(
        opDesc, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_NOT_PROPAGATE_NAN));

    // Perform the addition
    const float alpha = 1.0f;
    const float beta = 0.0f;
    checkCudnnError(hipdnnOpTensor(desc->handle, opDesc, &alpha,
                                  tensorDesc, a, &alpha, tensorDesc, b,
                                  &beta, tensorDesc, c));

    // Clean up
    checkCudnnError(hipdnnDestroyOpTensorDescriptor(opDesc));
    checkCudnnError(hipdnnDestroyTensorDescriptor(tensorDesc));
}

infiniopStatus_t cudaAdd(AddCudaDescriptor_t desc,
                         void *workspace,
                         unsigned long int workspace_size,
                         void *c, void *a, void *b,
                         void *stream) {
    if (dtype_eq(desc->dtype, F16)) {
        add_nv_gpu_f16(desc, c, a, b, stream);
        return STATUS_SUCCESS;
    }

    return STATUS_BAD_TENSOR_DTYPE;
}
